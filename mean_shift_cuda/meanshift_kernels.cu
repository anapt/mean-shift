#include "hip/hip_runtime.h"
#include "meanshift_kernels.h"
#include <stdio.h>

__global__ void calculate_kernel_matrix_kernel(Matrix shifted_points, Matrix original_points,
    double deviation, Matrix kernel_matrix){
    // Each thread calculates one element of kernel_matrix
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row * kernel_matrix.width + col > kernel_matrix.width * kernel_matrix.height){
        return;
    }

    int dimensions = shifted_points.width;
    // calculate distance
    double sum = 0, dif;
    for (int i=0; i<dimensions; i++){
        dif = shifted_points.elements[row * dimensions + i] - original_points.elements[col * dimensions + i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);

    double deviation_square = deviation*deviation;
    if (distance < deviation_square){
        // computes kernel matrix
        double pow = ((-1)*(distance * distance))/(2*(deviation_square));
        kernel_matrix.elements[row * kernel_matrix.width + col] = exp(pow);
    } else {
        kernel_matrix.elements[row * kernel_matrix.width + col] = 0;
    }
    if (row == col){
        kernel_matrix.elements[row * kernel_matrix.width + col] += 1;
    }
}

__global__ void shift_points_kernel(Matrix original_points, Matrix kernel_matrix, Matrix shifted_points,
    Matrix new_shift, Matrix denominator, Matrix mean_shift_vector){
    // Each thread computes one element of new_shift
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // performs calculations only if indexes are within matrix bounds
    //if (row + col < new_shift.height * new_shift.width){
    if (row < new_shift.height){
        // calculates new_shift
        // builds nominator by multiplying kernel_matrix and original_points
        for (int element_index = 0; element_index < kernel_matrix.width; ++element_index){
            cell_value += kernel_matrix.elements[row * kernel_matrix.width + element_index]
                * original_points.elements[element_index * original_points.width + col];
        }
        // new_shift elements are calculated by dividing with the denominator
        new_shift.elements[row * new_shift.width + col] =
            cell_value / denominator.elements[row];

        // calculates mean-shift vector
        mean_shift_vector.elements[row * new_shift.width + col] =
            new_shift.elements[row * new_shift.width + col] -
            shifted_points.elements[row * new_shift.width + col];
    }
}

__global__ void denominator_kernel(Matrix denominator, Matrix kernel_matrix){
    // Each thread computes one element of denominator_kernel
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < denominator.height){
        for (int column = 0; column < kernel_matrix.width; ++column){
             cell_value += kernel_matrix.elements[row * kernel_matrix.width + column];
        }
        denominator.elements[row] = cell_value;
    }
}