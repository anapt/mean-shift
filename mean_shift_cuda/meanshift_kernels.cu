#include "hip/hip_runtime.h"
#include "meanshift_kernels.h"
#include <stdio.h>

__global__ void multiply_kernel(Matrix matrix1, Matrix matrix2, Matrix output){
    // Each thread computes one element of output
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row + col < output.height * output.width){
        for (int element_index = 0; element_index < matrix1.width; ++element_index){
            cell_value += matrix1.elements[row * matrix1.width + element_index]
                * matrix2.elements[element_index * matrix2.width + col];
        }
        output.elements[row * output.width + col] = cell_value;
    }
}

__global__ void calculate_kernel_matrix_kernel(Matrix shifted_points, Matrix original_points
    , double deviation, Matrix kernel_matrix){
    // Each thread calculates one element of kernel_matrix
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row * kernel_matrix.width + col > kernel_matrix.width * kernel_matrix.height){
        return;
    }

    int dimensions = shifted_points.width;
    // calculate distance
    double sum = 0, dif;
    for (int i=0; i<dimensions; i++){
        dif = shifted_points.elements[row * dimensions + i] - original_points.elements[col * dimensions + i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);

    double deviation_square = deviation*deviation;
    if (distance < deviation_square){
        // computes kernel matrix
        double pow = ((-1)*(distance * distance))/(2*(deviation_square));
        kernel_matrix.elements[row * kernel_matrix.width + col] = exp(pow);
    } else {
        kernel_matrix.elements[row * kernel_matrix.width + col] = 0;
    }
    if (row == col){
        kernel_matrix.elements[row * kernel_matrix.width + col] += 1;
    }
}

__global__ void denominator_kernel<<<dimGrid, dimBlock>>>(Matrix denominator, Matrix kernel_matrix, int total){

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;


    if (row * denominator.width + col > denominator.width * denominator.height){
        return;
    }

    denominator[col]=0;
    denominator[row] += kernel_matrix[row*denominator.width + col];

}

// serial

// calculate denominator
for (int i=0; i<NUMBER_OF_POINTS; i++){
    double sum = 0;
    for (int j=0; j<NUMBER_OF_POINTS; j++){
        sum = sum + kernel_matrix[i][j];
    }
    denominator[i] = sum;
}