#include "hip/hip_runtime.h"
#include "meanshift_kernels.h"
#include <stdio.h>

__global__ void calculate_kernel_matrix_kernel(Matrix shifted_points, Matrix original_points,
    double deviation, Matrix kernel_matrix){
    // each thread calculates one element of kernel_matrix
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row * kernel_matrix.width + col >= kernel_matrix.width * kernel_matrix.height){
        return;
    }

    int dimensions = shifted_points.width;
    // calculate distance
    double sum = 0, dif;
    for (int i=0; i<dimensions; i++){
        dif = shifted_points.elements[row * dimensions + i]
            - original_points.elements[col * dimensions + i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);

    double deviation_square = deviation*deviation;
    if (distance < deviation_square){
        // computes kernel matrix
        double pow = ((-1)*(distance * distance))/(2*(deviation_square));
        kernel_matrix.elements[row * kernel_matrix.width + col] = exp(pow);
    } else {
        kernel_matrix.elements[row * kernel_matrix.width + col] = 0;
    }
    if (row == col){
        kernel_matrix.elements[row * kernel_matrix.width + col] += 1;
    }
}

__global__ void denominator_kernel(Matrix denominator, Matrix kernel_matrix){
    // each thread computes one element of denominator_kernel
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row >= denominator.height){
        return;
    }

    for (int column = 0; column < kernel_matrix.width; ++column){
         cell_value += kernel_matrix.elements[row * kernel_matrix.width + column];
    }
    denominator.elements[row] = cell_value;
}

__global__ void shift_points_kernel(Matrix original_points, Matrix kernel_matrix,
    Matrix shifted_points, Matrix new_shift, Matrix denominator, Matrix mean_shift_vector){
    // each thread computes one element of new_shift
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row * new_shift.width + col >= new_shift.width * new_shift.height){
        return;
    }

    // calculates new_shift
    // builds nominator by multiplying kernel_matrix and original_points
    for (int element_index = 0; element_index < kernel_matrix.width; ++element_index){
        cell_value += kernel_matrix.elements[row * kernel_matrix.width + element_index]
            * original_points.elements[element_index * original_points.width + col];
    }
    // new_shift elements are calculated by dividing with the denominator
    new_shift.elements[row * new_shift.width + col] =
        cell_value / denominator.elements[row];

    // calculates mean-shift vector
    mean_shift_vector.elements[row * new_shift.width + col] =
        new_shift.elements[row * new_shift.width + col] -
        shifted_points.elements[row * new_shift.width + col];
}