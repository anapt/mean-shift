#include "hip/hip_runtime.h"
#include "meanshift_kernels.h"
#include <stdio.h>

__global__ void calculate_kernel_matrix_kernel(Matrix shifted_points, Matrix original_points,
    double deviation, Matrix kernel_matrix){
    // each thread calculates one element of kernel_matrix
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row * kernel_matrix.width + col >= kernel_matrix.width * kernel_matrix.height){
        return;
    }

    int dimensions = shifted_points.width;
    // calculate distance
    double sum = 0, dif;
    for (int i=0; i<dimensions; i++){
        dif = shifted_points.elements[row * dimensions + i]
            - original_points.elements[col * dimensions + i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);

    double deviation_square = deviation*deviation;
    if (distance < deviation_square){
        // computes kernel matrix
        double pow = ((-1)*(distance * distance))/(2*(deviation_square));
        kernel_matrix.elements[row * kernel_matrix.width + col] = exp(pow);
    } else {
        kernel_matrix.elements[row * kernel_matrix.width + col] = 0;
    }
    if (row == col){
        kernel_matrix.elements[row * kernel_matrix.width + col] += 1;
    }
}

__global__ void denominator_kernel(Matrix denominator, Matrix kernel_matrix){
    // each thread computes one element of denominator_kernel
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row >= denominator.height){
        return;
    }

    for (int column = 0; column < kernel_matrix.width; ++column){
         cell_value += kernel_matrix.elements[row * kernel_matrix.width + column];
    }
    denominator.elements[row] = cell_value;
}

__global__ void shift_points_kernel(Matrix original_points, Matrix kernel_matrix,
    Matrix shifted_points, Matrix new_shift, Matrix denominator, Matrix mean_shift_vector){
    // each thread computes one element of new_shift
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row * new_shift.width + col >= new_shift.width * new_shift.height){
        return;
    }

    // calculates new_shift
    // builds nominator by multiplying kernel_matrix and original_points
    for (int element_index = 0; element_index < kernel_matrix.width; ++element_index){
        cell_value += kernel_matrix.elements[row * kernel_matrix.width + element_index]
            * original_points.elements[element_index * original_points.width + col];
    }
    // new_shift elements are calculated by dividing with the denominator
    new_shift.elements[row * new_shift.width + col] =
        cell_value / denominator.elements[row];

    // calculates mean-shift vector
    mean_shift_vector.elements[row * new_shift.width + col] =
        new_shift.elements[row * new_shift.width + col] -
        shifted_points.elements[row * new_shift.width + col];
}

__global__ void norm(Matrix mean_shift_vector, double *current_norm) {
    // each thread computes one element of new_shift
    // by accumulating results into cell_value
    double cell_value = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // performs calculations only if thread's indexes are within matrix bounds
    if (row * mean_shift_vector.width + col >= mean_shift_vector.width * mean_shift_vector.height){
        return;
    }

    for (int element_index = 0; element_index < mean_shift_vector.width; ++element_index){
        cell_value += mean_shift_vector.elements[row * mean_shift_vector.width + element_index]
                      * mean_shift_vector.elements[row * mean_shift_vector.width + element_index];
    }

    &current_norm = sqrt(cell_value);


//    // new_shift elements are calculated by dividing with the denominator
//    new_shift.elements[row * new_shift.width + col] =
//            cell_value / denominator.elements[row];
//
//    // calculates mean-shift vector
//    mean_shift_vector.elements[row * new_shift.width + col] =
//            new_shift.elements[row * new_shift.width + col] -
//            shifted_points.elements[row * new_shift.width + col];

//    int n_tid = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
//    int i = 1;
//    int initial_tid = n_tid / 2;
//    int limit = gridDim.x * blockDim.x;

//    int block_end = 2 * (blockIdx.x * blockDim.x + blockDim.x) - 1;
//
//    if (n_tid < (2 * limit)){
//
//        while ( (i < (2 * blockDim.x)) && n_tid < block_end &&
//                (n_tid + i) <= block_end){
//
//            norms[n_tid] += norms[n_tid + i];
//            n_tid = n_tid + i * (initial_tid * 2 - 2 * (blockIdx.x * blockDim.x));
//            i *= 2;
//            __syncthreads();
//        }
//
//
//        if (!((initial_tid) % blockDim.x))
//            norm_per_block[blockIdx.x] = norms[n_tid];
//
//    }
}