#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "meanshift_utils.h"
#include "meanshift_kernels.h"

#define OUTPUT_PREFIX "../output/output_"

hipDeviceProp_t device_properties;

void get_args(int argc, char **argv, parameters *params){
    if (argc < 7) {
        printf("Usage: %s h e N D Pd Pl\nwhere:\n"
        "\th is the variance\n"
        "\te is the min distance, between two points, that is taken into account in computations\n"
        "\tN is the the number of points\n"
        "\tD is the number of dimensions of each point\n"
        "\tPd is the path of the dataset file\n"
        "\tPl is the path of the labels file\n"
        "\n\t--verbose | -v is an optional flag to enable execution information output"
        "\n\t--output | -o is an optional flag to enable points output in each iteration", argv[0]);
        exit(1);
    }

    DEVIATION = atoi(argv[1]);
    params->epsilon = atof(argv[2]);
    NUMBER_OF_POINTS = atoi(argv[3]);
    DIMENSIONS = atoi(argv[4]);
    POINTS_FILENAME = argv[5];
    LABELS_FILENAME = argv[6];
    params->verbose = false;
    params->display = false;

    if (argc > 7){
        for (int index=7; index<argc; ++index){
            if (!strcmp(argv[index], "--verbose") || !strcmp(argv[index], "-v")){
                params->verbose = true;
            } else if (!strcmp(argv[index], "--output") || !strcmp(argv[index], "-o")){
                params->display = true;
            } else {
                printf("Couldn't parse argument %d: %s\n", index, argv[index]);
                exit(EXIT_FAILURE);
            }
        }
    }

    /*printf("DEVIATION = %d\n"
        "epsilon = %f\n"
        "NUMBER_OF_POINTS = %d\n"
        "DIMENSIONS = %d\n"
        "POINTS_FILENAME = %s\n"
        "LABELS_FILENAME = %s\n"
        "verbose = %d\n"
        "display = %d\n", DEVIATION, params->epsilon, NUMBER_OF_POINTS, DIMENSIONS, POINTS_FILENAME
            , LABELS_FILENAME, params->verbose, params->display);*/
}

void init(double ***vectors, char **labels){
    int bytes_read = 0;

    set_Gpu();

    if (params.verbose){
        printf("Reading dataset and labels...\n");
    }

    // initializes vectors
    FILE *points_file;
    points_file = fopen(POINTS_FILENAME, "rb");
    if (points_file != NULL){
        // allocates memory for the array
        (*vectors) = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        // reads vectors dataset from file
        for (int i=0; i<NUMBER_OF_POINTS; i++){
            bytes_read = fread((*vectors)[i], sizeof(double), DIMENSIONS, points_file);
            if ( bytes_read != DIMENSIONS ){
                if(feof(points_file)){
                    printf("Premature end of file reached.\n");
                } else{
                    printf("Error reading points file.");
                }
                fclose(points_file);
                exit(EXIT_FAILURE);
            }
        }
    } else {
        printf("Error reading dataset file.\n");
        exit(EXIT_FAILURE);
    }
    fclose(points_file);

    // initializes file that will contain the labels (train)
    FILE *labels_file;
    labels_file = fopen(LABELS_FILENAME, "rb");
    if (labels_file != NULL){
        // NOTE : Labels were classified as <class 'numpy.uint8'>
        // variables of type uint8 are stored as 1-byte (8-bit) unsigned integers
        // gets number of labels
        fseek(labels_file, 0L, SEEK_END);
        long int pos = ftell(labels_file);
        rewind(labels_file);
        int label_elements = pos/ sizeof(char);

        // allocates memory for the array
        *labels = (char*)malloc(label_elements* sizeof(char));
        fseek(labels_file, 0L, SEEK_SET);
        bytes_read = fread((*labels), sizeof(char), label_elements, labels_file);
        if ( bytes_read != label_elements ){
            if(feof(points_file)){
                printf("Premature end of file reached.\n");
            } else{
                printf("Error reading points file.");
            }
            fclose(labels_file);
            exit(EXIT_FAILURE);
        }
    }
    fclose(labels_file);

    if (params.verbose){
        printf("Done.\n\n");
    }
}

//Based on https://stackoverflow.com/a/28113186
//Poio psagmeno link https://www.cs.virginia.edu/~csadmin/wiki/index.php/CUDA_Support/Choosing_a_GPU
void set_Gpu(){
    int devices_count = 0, max_multiprocessors = 0, max_device = 0;

    // gets devices count checking for errors like no devices or no drivers to check for
    // devices available
    gpuErrchk( hipGetDeviceCount(&devices_count) );
    for(int device_index = 0; device_index < devices_count; ++device_index){
        // gets current index device's properties
        hipDeviceProp_t this_device_properties;
        gpuErrchk( hipGetDeviceProperties(&this_device_properties, device_index) );

        // stores best available device's index
        // only devices with compute capability >= 2.0 are able to run the code
        if (max_multiprocessors < this_device_properties.multiProcessorCount
            && this_device_properties.major >= 2 && this_device_properties.minor >= 0){
            // stores devices properties for later use
            device_properties = this_device_properties;
            max_multiprocessors = this_device_properties.multiProcessorCount;
            max_device = device_index;
        }
    }
    // sets the device
    gpuErrchk( hipSetDevice(max_device) );
    if (params.verbose){
        printf("Device chosen is \"%s\"\n"
            "Device has %d multi processors and compute capability %d.%d\n"
            "Max threads per block supported are %d\n\n"
            , device_properties.name
            , device_properties.multiProcessorCount, device_properties.major, device_properties.minor
            , device_properties.maxThreadsPerBlock);
    }
}

int meanshift(double **original_points, double ***shifted_points, int deviation
    , parameters *opt){
    static int iteration = 0;
    static double **mean_shift_vector, **kernel_matrix, *denominator;

    // allocates memory and copies original points on first iteration
    if (iteration == 0 || (*shifted_points) == NULL){
        (*shifted_points) = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        duplicate(original_points, NUMBER_OF_POINTS, DIMENSIONS, shifted_points);

        // allocates memory for mean shift vector
        mean_shift_vector = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        // initializes elements of mean_shift_vector to inf
        for (int i=0;i<NUMBER_OF_POINTS;i++){
            for (int j=0;j<DIMENSIONS;j++){
                mean_shift_vector[i][j] = DBL_MAX;
            }
        }

        // allocates memory for other arrays needed
        kernel_matrix = alloc_2d_double(NUMBER_OF_POINTS, NUMBER_OF_POINTS);
        denominator = (double *)malloc(NUMBER_OF_POINTS * sizeof(double));
    }

    // finds pairwise distance matrix (inside radius)
    // [I, D] = rangesearch(x,y,h);
    calculate_kernel_matrix((*shifted_points), original_points, deviation, &kernel_matrix);
    // calculate denominator
    for (int i=0; i<NUMBER_OF_POINTS; i++){
        double sum = 0;
        for (int j=0; j<NUMBER_OF_POINTS; j++){
            sum = sum + kernel_matrix[i][j];
        }
        denominator[i] = sum;
    }

    // creates new y vector
    double **new_shift = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);

    // builds nominator
    multiply(kernel_matrix, original_points, &new_shift);

    // divides element-wise
    for (int i=0; i<NUMBER_OF_POINTS; i++){
        for (int j=0; j<DIMENSIONS; j++){
            new_shift[i][j] = new_shift[i][j] / denominator[i];
            // calculates mean-shift vector at the same time
            mean_shift_vector[i][j] = new_shift[i][j] - (*shifted_points)[i][j];
        }
    }

    // frees previously shifted points, they're now garbage
    free((*shifted_points)[0]);
    // updates shifted points pointer to the new array address
    shifted_points = &new_shift;

    if (params.display){
        save_matrix((*shifted_points), iteration);
    }

    // calculates norm of the new mean shift vector
    double current_norm = norm(mean_shift_vector, NUMBER_OF_POINTS, DIMENSIONS);
    if (params.verbose){
        printf("Iteration n. %d, error %f \n", iteration, current_norm);
    }

    /** iterates until convergence **/
    if (current_norm > opt->epsilon) {
        ++iteration;
        meanshift(original_points, shifted_points, deviation, opt);
    }

    if (iteration == 0){
        // cleans up allocations
        free(mean_shift_vector[0]);
        free(mean_shift_vector);
        free(kernel_matrix[0]);
        free(kernel_matrix);
        free(denominator);
    }

    return iteration;
}

// TODO check why there's is a difference in the norm calculate in matlab
double norm(double **matrix, int rows, int cols){
    double sum=0, temp_mul=0;
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            temp_mul = matrix[i][j] * matrix[i][j];
            sum = sum + temp_mul;
        }
    }
    double norm = sqrt(sum);
    return norm;
}

void calculate_kernel_matrix(double **shifted_points, double **original_points, double deviation
    , double ***kernel_matrix){
    static bool first_iter = true;

    // allocates memory for shifted_points in GPU and copies the array
    Matrix d_shifted_points;
    d_shifted_points.width = DIMENSIONS;
    d_shifted_points.height = NUMBER_OF_POINTS;
    int size = DIMENSIONS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&d_shifted_points.elements, size) );
    gpuErrchk( hipMemcpy(d_shifted_points.elements, &(shifted_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for original_points in GPU and copies the array
    Matrix d_original_points;
    d_original_points.width = DIMENSIONS;
    d_original_points.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&d_original_points.elements, size) );
    gpuErrchk( hipMemcpy(d_original_points.elements, &(original_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for kernel_matrix in GPU
    Matrix d_kernel_matrix;
    d_kernel_matrix.width = NUMBER_OF_POINTS;
    d_kernel_matrix.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&d_kernel_matrix.elements, size) );

    // get max sizes supported from the device
    int max_block_size = (int)sqrt(device_properties.maxThreadsPerBlock);
    int requested_block_size = max_block_size;
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;

    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = requested_block_size;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = (d_kernel_matrix.width + dimBlock.y - 1) / dimBlock.y;

        calculate_kernel_matrix_kernel<<<dimGrid, dimBlock>>>(d_shifted_points, d_original_points
            , deviation, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_kernel_matrix_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMemcpy(&((*kernel_matrix)[0][0]), d_kernel_matrix.elements
        , size, hipMemcpyDeviceToHost) );

    gpuErrchk( hipFree(d_shifted_points.elements) );
    gpuErrchk( hipFree(d_original_points.elements) );
    gpuErrchk( hipFree(d_kernel_matrix.elements) );
}


void multiply(double **kernel_matrix, double **original_points, double ***new_shift){
    static bool first_iter = true;

    // allocates memory for kernel_matrix in GPU and copies the array
    Matrix d_kernel_matrix;
    d_kernel_matrix.width = NUMBER_OF_POINTS;
    d_kernel_matrix.height = NUMBER_OF_POINTS;
    int size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&d_kernel_matrix.elements, size) );
    gpuErrchk( hipMemcpy(d_kernel_matrix.elements, &(kernel_matrix[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for original_points in GPU and copies the array
    Matrix d_original_points;
    d_original_points.width = DIMENSIONS;
    d_original_points.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&d_original_points.elements, size) );
    gpuErrchk( hipMemcpy(d_original_points.elements, &(original_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for new_shift in GPU
    Matrix d_new_shift;
    d_new_shift.width = DIMENSIONS;
    d_new_shift.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&d_new_shift.elements, size) );

    // get max sizes supported from the device
    int max_block_size = device_properties.maxThreadsPerBlock;
    dim3 dimBlock((d_new_shift.height < sqrt(max_block_size)) ? d_new_shift.height : sqrt(max_block_size)
        , (d_new_shift.width < sqrt(max_block_size)) ? d_new_shift.width : sqrt(max_block_size));
    dim3 dimGrid((d_new_shift.height + dimBlock.x - 1) / dimBlock.x
        , (d_new_shift.width + dimBlock.y - 1) / dimBlock.y);

    if (first_iter && params.verbose){
        printf("multiply_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    multiply_kernel<<<dimGrid, dimBlock>>>(d_kernel_matrix, d_original_points, d_new_shift);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMemcpy(&((*new_shift)[0][0]), d_new_shift.elements
        , size, hipMemcpyDeviceToHost) );

    gpuErrchk( hipFree(d_kernel_matrix.elements) );
    gpuErrchk( hipFree(d_original_points.elements) );
    gpuErrchk( hipFree(d_new_shift.elements) );
}

double calculateDistance(double *y, double *x){
    double sum = 0, dif;
    for (int i=0; i<DIMENSIONS; i++){
        dif = y[i]-x[i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);
    return distance;
}

double **alloc_2d_double(int rows, int cols) {
    double *data = (double *) malloc(rows*cols*sizeof(double));
    double **array = (double **) malloc(rows*sizeof(double*));
    for (int i=0; i<rows; i++)
        array[i] = &(data[cols*i]);
    return array;
}

void duplicate(double **source, int rows, int cols, double ***dest){
    for (int i=0; i<rows; i++){
        for (int j=0; j<cols; j++){
            (*dest)[i][j] = source[i][j];
        }
    }
}

void print_matrix(double **array, int rows, int cols){
    for (int i=0; i<cols; i++){
        for (int j=0; j<rows; j++){
            printf("%f ", array[j][i]);
        }
        printf("\n");
    }
}

void save_matrix(double **matrix, int iteration){
    char filename[50];
    snprintf(filename, sizeof(filename), "%s%d", "../output/output_", iteration);
    FILE *file;
    file = fopen(filename, "w");
    for (int rows=0; rows<NUMBER_OF_POINTS; ++rows){
        for (int cols=0; cols<DIMENSIONS; ++cols){
            fprintf(file, "%f", matrix[rows][cols]);
            if (cols != DIMENSIONS - 1){
                fprintf(file, ",");
            }
        }
        fprintf(file, "\n");
    }
}
