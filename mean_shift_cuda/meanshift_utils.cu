#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "meanshift_utils.h"
#include "meanshift_kernels.h"

#define OUTPUT_PREFIX "../output/output_"
#define BLOCK_SIZE 8

void get_args(int argc, char **argv){
    if (argc != 6) {
        printf("Usage: %s h N D Pd Pl\nwhere:\n", argv[0]);
        printf("\th is the variance\n");
        printf("\tN is the the number of points\n");
        printf("\tD is the number of dimensions of each point\n");
        printf("\tPd is the path of the dataset file\n");
        printf("\tPl is the path of the labels file\n");
        exit(1);
    }

    DEVIATION = atoi(argv[1]);
    NUMBER_OF_POINTS = atoi(argv[2]);
    DIMENSIONS = atoi(argv[3]);
    POINTS_FILENAME = argv[4];
    LABELS_FILENAME = argv[5];
}

void init(double ***vectors, char **labels, parameters *params){
    int bytes_read = 0;
    // initializes vectors
    FILE *points_file;
    points_file = fopen(POINTS_FILENAME, "rb");
    if (points_file != NULL){
        // allocates memory for the array
        (*vectors) = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        // reads vectors dataset from file
        for (int i=0; i<NUMBER_OF_POINTS; i++){
            bytes_read = fread((*vectors)[i], sizeof(double), DIMENSIONS, points_file);
            if ( bytes_read != DIMENSIONS ){
                if(feof(points_file)){
                    printf("Premature end of file reached.\n");
                } else{
                    printf("Error reading points file.");
                }
                fclose(points_file);
                exit(EXIT_FAILURE);
            }
        }
    } else {
        printf("Error reading dataset file.\n");
        exit(EXIT_FAILURE);
    }
    fclose(points_file);

    // initializes file that will contain the labels (train)
    FILE *labels_file;
    labels_file = fopen(LABELS_FILENAME, "rb");
    if (labels_file != NULL){
        // NOTE : Labels were classified as <class 'numpy.uint8'>
        // variables of type uint8 are stored as 1-byte (8-bit) unsigned integers
        // gets number of labels
        fseek(labels_file, 0L, SEEK_END);
        long int pos = ftell(labels_file);
        rewind(labels_file);
        int label_elements = pos/ sizeof(char);

        // allocates memory for the array
        *labels = (char*)malloc(label_elements* sizeof(char));
        fseek(labels_file, 0L, SEEK_SET);
        bytes_read = fread((*labels), sizeof(char), label_elements, labels_file);
        if ( bytes_read != label_elements ){
            if(feof(points_file)){
                printf("Premature end of file reached.\n");
            } else{
                printf("Error reading points file.");
            }
            fclose(labels_file);
            exit(EXIT_FAILURE);
        }
    }
    fclose(labels_file);

    // MEAN SHIFT OPTIONS
    params->epsilon = 0.0001;
    params->verbose = false;
    params->display = false;
}

int meanshift(double **original_points, double ***shifted_points, int deviation
    , parameters *opt){
    static int iteration = 0;
    static double **mean_shift_vector, **kernel_matrix, *denominator;

    // allocates memory and copies original points on first iteration
    if (iteration == 0 || (*shifted_points) == NULL){
        (*shifted_points) = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        duplicate(original_points, NUMBER_OF_POINTS, DIMENSIONS, shifted_points);

        // allocates memory for mean shift vector
        mean_shift_vector = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        // initializes elements of mean_shift_vector to inf
        for (int i=0;i<NUMBER_OF_POINTS;i++){
            for (int j=0;j<DIMENSIONS;j++){
                mean_shift_vector[i][j] = DBL_MAX;
            }
        }

        // allocates memory for other arrays needed
        kernel_matrix = alloc_2d_double(NUMBER_OF_POINTS, NUMBER_OF_POINTS);
        denominator = (double *)malloc(NUMBER_OF_POINTS * sizeof(double));
    }

    // finds pairwise distance matrix (inside radius)
    // [I, D] = rangesearch(x,y,h);
    for (int i=0; i<NUMBER_OF_POINTS; i++){
        double sum = 0;
        for (int j=0; j<NUMBER_OF_POINTS; j++){
            double distance = calculateDistance((*shifted_points)[i]
                , original_points[j]);

            double deviation_square = deviation*deviation;
            if (distance < deviation_square){
                // computes kernel matrix
                double pow = ((-1)*(distance * distance))/(2*(deviation_square));
                kernel_matrix[i][j] = exp(pow);
            } else {
                kernel_matrix[i][j] = 0;
            }
            if (i == j){
                kernel_matrix[i][j] += 1;
            }
            sum = sum + kernel_matrix[i][j];
        }
        denominator[i] = sum;
    }

    // creates new y vector
    double **new_shift = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);

    // builds nominator
    multiply(kernel_matrix, original_points, &new_shift);

    // divides element-wise
    for (int i=0; i<NUMBER_OF_POINTS; i++){
        for (int j=0; j<DIMENSIONS; j++){
            new_shift[i][j] = new_shift[i][j] / denominator[i];
            // calculates mean-shift vector at the same time
            mean_shift_vector[i][j] = new_shift[i][j] - (*shifted_points)[i][j];
        }
    }

    // frees previously shifted points, they're now garbage
    free((*shifted_points)[0]);
    // updates shifted points pointer to the new array address
    shifted_points = &new_shift;

    save_matrix((*shifted_points), iteration);

    // calculates norm of the new mean shift vector
    double current_norm = norm(mean_shift_vector, NUMBER_OF_POINTS, DIMENSIONS);
    printf("Iteration n. %d, error %f \n", iteration, current_norm);

    /** iterates until convergence **/
    if (current_norm > opt->epsilon) {
        ++iteration;
        meanshift(original_points, shifted_points, deviation, opt);
    }

    if (iteration == 0){
        // cleans up this iteration's allocations
        free(mean_shift_vector[0]);
        free(mean_shift_vector);
        free(kernel_matrix[0]);
        free(kernel_matrix);
        free(denominator);
    }

    return iteration;
}

// TODO check why there's is a difference in the norm calculate in matlab
double norm(double **matrix, int rows, int cols){
    double sum=0, temp_mul=0;
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            temp_mul = matrix[i][j] * matrix[i][j];
            sum = sum + temp_mul;
        }
    }
    double norm = sqrt(sum);
    return norm;
}

void multiply(double **kernel_matrix, double **original_points, double ***new_shift){
	// allocates memory for kernel_matrix in GPU and copies the array
	Matrix d_kernel_matrix;
    d_kernel_matrix.width = NUMBER_OF_POINTS;
    d_kernel_matrix.height = NUMBER_OF_POINTS;
    int size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&d_kernel_matrix.elements, size) );
    gpuErrchk( hipMemcpy(d_kernel_matrix.elements, &(kernel_matrix[0][0])
    	, size, hipMemcpyHostToDevice) );

    // allocates memory for original_points in GPU and copies the array
    Matrix d_original_points;
    d_original_points.width = DIMENSIONS;
    d_original_points.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&d_original_points.elements, size) );
    gpuErrchk( hipMemcpy(d_original_points.elements, &(original_points[0][0])
    	, size, hipMemcpyHostToDevice) );

	// allocates memory for new_shift in GPU
    Matrix d_new_shift;
    d_new_shift.width = DIMENSIONS;
    d_new_shift.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&d_new_shift.elements, size) );

    //dim3 dimBlock(16, 16);
    //dim3 dimGrid(d_original_points.width / dimBlock.x, d_kernel_matrix.height / dimBlock.y);

    dim3 dimBlock(10, 2);
    dim3 dimGrid(60, 1);

    multiply_kernel<<<dimGrid, dimBlock>>>(d_kernel_matrix, d_original_points, d_new_shift);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMemcpy(&((*new_shift)[0][0]), d_new_shift.elements
    	, size, hipMemcpyDeviceToHost) );

    gpuErrchk( hipFree(d_kernel_matrix.elements) );
    gpuErrchk( hipFree(d_original_points.elements) );
    gpuErrchk( hipFree(d_new_shift.elements) );
}

double calculateDistance(double *y, double *x){
    double sum = 0, dif;
    for (int i=0; i<DIMENSIONS; i++){
        dif = y[i]-x[i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);
    return distance;
}

double **alloc_2d_double(int rows, int cols) {
    double *data = (double *) malloc(rows*cols*sizeof(double));
    double **array = (double **) malloc(rows*sizeof(double*));
    for (int i=0; i<rows; i++)
        array[i] = &(data[cols*i]);
    return array;
}

void duplicate(double **source, int rows, int cols, double ***dest){
    for (int i=0; i<rows; i++){
        for (int j=0; j<cols; j++){
            (*dest)[i][j] = source[i][j];
        }
    }
}

void print_matrix(double **array, int rows, int cols){
    for (int i=0; i<cols; i++){
        for (int j=0; j<rows; j++){
            printf("%f ", array[j][i]);
        }
        printf("\n");
    }
}

void save_matrix(double **matrix, int iteration){
    char filename[50];
    snprintf(filename, sizeof(filename), "%s%d", "../output/output_", iteration);
    FILE *file;
    file = fopen(filename, "w");
    for (int rows=0; rows<NUMBER_OF_POINTS; ++rows){
        for (int cols=0; cols<DIMENSIONS; ++cols){
            fprintf(file, "%f", matrix[rows][cols]);
            if (cols != DIMENSIONS - 1){
                fprintf(file, ",");
            }
        }
        fprintf(file, "\n");
    }
}
