#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>

#include "meanshift_utils.h"

#define OUTPUT_PREFIX "../output/output_"

hipDeviceProp_t device_properties;

void get_args(int argc, char **argv, parameters *params){
    if (argc < 7) {
        printf("Usage: %s h e N D Pd Pl\nwhere:\n"
        "\th is the variance\n"
        "\te is the min distance, between two points, that is taken into account in computations\n"
        "\tN is the the number of points\n"
        "\tD is the number of dimensions of each point\n"
        "\tPd is the path of the dataset file\n"
        "\tPl is the path of the labels file\n"
        "\n\t--verbose | -v is an optional flag to enable execution information output"
        "\n\t--output | -o is an optional flag to enable points output in each iteration", argv[0]);
        exit(1);
    }

    DEVIATION = atoi(argv[1]);
    params->epsilon = atof(argv[2]);
    NUMBER_OF_POINTS = atoi(argv[3]);
    DIMENSIONS = atoi(argv[4]);
    POINTS_FILENAME = argv[5];
    LABELS_FILENAME = argv[6];
    params->verbose = false;
    params->display = false;

    if (argc > 7){
        for (int index=7; index<argc; ++index){
            if (!strcmp(argv[index], "--verbose") || !strcmp(argv[index], "-v")){
                params->verbose = true;
            } else if (!strcmp(argv[index], "--output") || !strcmp(argv[index], "-o")){
                params->display = true;
            } else {
                printf("Couldn't parse argument %d: %s\n", index, argv[index]);
                exit(EXIT_FAILURE);
            }
        }
    }

    /*printf("DEVIATION = %d\n"
        "epsilon = %f\n"
        "NUMBER_OF_POINTS = %d\n"
        "DIMENSIONS = %d\n"
        "POINTS_FILENAME = %s\n"
        "LABELS_FILENAME = %s\n"
        "verbose = %d\n"
        "display = %d\n", DEVIATION, params->epsilon, NUMBER_OF_POINTS, DIMENSIONS, POINTS_FILENAME
            , LABELS_FILENAME, params->verbose, params->display);*/
}

void init(double ***vectors, char **labels){
    int bytes_read = 0;

    set_GPU();

    if (params.verbose){
        printf("Reading dataset and labels...\n");
    }

    // initializes vectors
    FILE *points_file;
    points_file = fopen(POINTS_FILENAME, "rb");
    if (points_file != NULL){
        // allocates memory for the array
        (*vectors) = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        // reads vectors dataset from file
        for (int i=0; i<NUMBER_OF_POINTS; i++){
            bytes_read = fread((*vectors)[i], sizeof(double), DIMENSIONS, points_file);
            if ( bytes_read != DIMENSIONS ){
                if(feof(points_file)){
                    printf("Premature end of file reached.\n");
                } else{
                    printf("Error reading points file.");
                }
                fclose(points_file);
                exit(EXIT_FAILURE);
            }
        }
    } else {
        printf("Error reading dataset file.\n");
        exit(EXIT_FAILURE);
    }
    fclose(points_file);

    // initializes file that will contain the labels (train)
    FILE *labels_file;
    labels_file = fopen(LABELS_FILENAME, "rb");
    if (labels_file != NULL){
        // NOTE : Labels were classified as <class 'numpy.uint8'>
        // variables of type uint8 are stored as 1-byte (8-bit) unsigned integers
        // gets number of labels
        fseek(labels_file, 0L, SEEK_END);
        long int pos = ftell(labels_file);
        rewind(labels_file);
        int label_elements = pos/ sizeof(char);

        // allocates memory for the array
        *labels = (char*)malloc(label_elements* sizeof(char));
        fseek(labels_file, 0L, SEEK_SET);
        bytes_read = fread((*labels), sizeof(char), label_elements, labels_file);
        if ( bytes_read != label_elements ){
            if(feof(points_file)){
                printf("Premature end of file reached.\n");
            } else{
                printf("Error reading points file.");
            }
            fclose(labels_file);
            exit(EXIT_FAILURE);
        }
    }
    fclose(labels_file);

    if (params.verbose){
        printf("Done.\n\n");
    }
}

//Based on https://stackoverflow.com/a/28113186
//Poio psagmeno link https://www.cs.virginia.edu/~csadmin/wiki/index.php/CUDA_Support/Choosing_a_GPU
void set_GPU(){
    int devices_count = 0, max_multiprocessors = 0, max_device = 0;

    // gets devices count checking for errors like no devices or no drivers to check for
    // devices available
    gpuErrchk( hipGetDeviceCount(&devices_count) );
    for(int device_index = 0; device_index < devices_count; ++device_index){
        // gets current index device's properties
        hipDeviceProp_t this_device_properties;
        gpuErrchk( hipGetDeviceProperties(&this_device_properties, device_index) );

        // stores best available device's index
        // only devices with compute capability >= 2.0 are able to run the code
        if (max_multiprocessors < this_device_properties.multiProcessorCount
            && this_device_properties.major >= 2 && this_device_properties.minor >= 0){
            // stores devices properties for later use
            device_properties = this_device_properties;
            max_multiprocessors = this_device_properties.multiProcessorCount;
            max_device = device_index;
        }
    }
    // sets the device
    gpuErrchk( hipSetDevice(max_device) );
    if (params.verbose){
        printf("Device chosen is \"%s\"\n"
            "Device has %d multi processors and compute capability %d.%d\n"
            "Max threads per block supported are %d\n\n"
            , device_properties.name
            , device_properties.multiProcessorCount, device_properties.major, device_properties.minor
            , device_properties.maxThreadsPerBlock);
    }
}

int meanshift(double **original_points, double ***shifted_points, int deviation
    , parameters *opt){
    // host variables
    int size = 0;
    static int iteration = 0;
    static double **kernel_matrix, *denominator, **mean_shift_vector;
    double **new_shift;

    // device variables
    static Matrix d_original_points, d_shifted_points, d_kernel_matrix, d_denominator,
        d_mean_shift_vector;
    Matrix d_new_shift;

    // allocates memory and copies original points on first iteration
    if (iteration == 0 || (*shifted_points) == NULL){
        // allocates memory for shifted points array and copies original points into it
        (*shifted_points) = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        duplicate(original_points, NUMBER_OF_POINTS, DIMENSIONS, shifted_points);

        // allocates memory for mean shift vector
        mean_shift_vector = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
        // initializes elements of mean_shift_vector to inf
        for (int i=0;i<NUMBER_OF_POINTS;i++){
            for (int j=0;j<DIMENSIONS;j++){
                mean_shift_vector[i][j] = DBL_MAX;
            }
        }

        // allocates memory for other arrays needed
        kernel_matrix = alloc_2d_double(NUMBER_OF_POINTS, NUMBER_OF_POINTS);
        denominator = (double *)malloc(NUMBER_OF_POINTS * sizeof(double));

        // allocates corresponding memory in device
        init_device_memory(original_points, *shifted_points, &d_original_points, &d_shifted_points,
            &d_kernel_matrix, &d_denominator, &d_mean_shift_vector);
    }

    // finds pairwise distance matrix (inside radius)
    // [I, D] = rangesearch(x,y,h);
    calculate_kernel_matrix(d_shifted_points, d_original_points, d_kernel_matrix, deviation,
        &kernel_matrix);

    // calculates denominator
    calculate_denominator(d_kernel_matrix, d_denominator, &denominator);

    size = NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMemcpy(d_denominator.elements, &(denominator[0])
        , size, hipMemcpyHostToDevice) );

    // creates new y vector
    // allocates memory in every recursion
    new_shift = alloc_2d_double(NUMBER_OF_POINTS, DIMENSIONS);
    // allocates corresponding memory in device
    d_new_shift.width = DIMENSIONS;
    d_new_shift.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_new_shift.elements), size) );

    shift_points(d_kernel_matrix, d_original_points, d_shifted_points, d_new_shift, d_denominator,
        d_mean_shift_vector, kernel_matrix, original_points, &new_shift, &mean_shift_vector);

    // frees previously shifted points, they're now garbage
    free((*shifted_points)[0]);
    // updates shifted points pointer to the new array address
    shifted_points = &new_shift;
    d_shifted_points.elements = d_new_shift.elements;

    if (params.display){
        save_matrix((*shifted_points), iteration);
    }

    // calculates norm of the new mean shift vector
    double current_norm = norm(mean_shift_vector, NUMBER_OF_POINTS, DIMENSIONS);
    if (params.verbose){
        printf("Iteration n. %d, error %f \n", iteration, current_norm);
    }

    /** iterates until convergence **/
    if (current_norm > opt->epsilon) {
        ++iteration;
        meanshift(original_points, shifted_points, deviation, opt);
    }

    if (iteration == 0){
        // cleans up allocations
        free(mean_shift_vector[0]);
        free(mean_shift_vector);
        free(kernel_matrix[0]);
        free(kernel_matrix);
        free(denominator);

        free_device_memory(d_original_points, d_kernel_matrix, d_denominator, d_new_shift);
    }

    return iteration;
}

// TODO check why there's is a difference in the norm calculate in matlab
double norm(double **matrix, int rows, int cols){
    double sum=0, temp_mul=0;
    for (int i=0; i<rows; i++) {
        for (int j=0; j<cols; j++) {
            temp_mul = matrix[i][j] * matrix[i][j];
            sum = sum + temp_mul;
        }
    }
    double norm = sqrt(sum);
    return norm;
}

void init_device_memory(double **original_points, double **shifted_points,
    Matrix *d_original_points, Matrix *d_shifted_points, Matrix *d_kernel_matrix,
    Matrix *d_denominator, Matrix *d_mean_shift_vector){
    int size;

    // allocates memory for original_points in GPU and copies the array
    d_original_points->width = DIMENSIONS;
    d_original_points->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_original_points->elements), size) );
    gpuErrchk( hipMemcpy(d_original_points->elements, &(original_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for shifted_points in GPU and copies the array
    d_shifted_points->width = DIMENSIONS;
    d_shifted_points->height = NUMBER_OF_POINTS;
    size = DIMENSIONS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_shifted_points->elements), size) );
    gpuErrchk( hipMemcpy(d_shifted_points->elements, &(shifted_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for kernel_matrix in GPU
    d_kernel_matrix->width = NUMBER_OF_POINTS;
    d_kernel_matrix->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_kernel_matrix->elements), size) );

    // allocates memory for denominator in GPU
    d_denominator->width = 1;
    d_denominator->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_denominator->elements), size) );

    // allocates memory for mean_shift_vector in GPU
    d_mean_shift_vector->width = DIMENSIONS;
    d_mean_shift_vector->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_mean_shift_vector->elements), size) );
}

void calculate_kernel_matrix(Matrix d_shifted_points, Matrix d_original_points,
    Matrix d_kernel_matrix, double deviation, double ***kernel_matrix){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)sqrt(max_block_size);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = requested_block_size;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = (d_kernel_matrix.width + dimBlock.y - 1) / dimBlock.y;

        calculate_kernel_matrix_kernel<<<dimGrid, dimBlock>>>(d_shifted_points, d_original_points
            , deviation, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_kernel_matrix_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMemcpy(&((*kernel_matrix)[0][0]), d_kernel_matrix.elements
        , size, hipMemcpyDeviceToHost) );
}

void calculate_denominator(Matrix d_kernel_matrix, Matrix d_denominator, double **denominator){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int requested_block_size = device_properties.maxThreadsPerBlock;
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = 1;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        denominator_kernel<<<dimGrid, dimBlock>>>(d_denominator, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_denominator called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMemcpy(&((*denominator)[0]), d_denominator.elements
    	, size, hipMemcpyDeviceToHost) );
}

void shift_points(Matrix d_kernel_matrix, Matrix d_original_points, Matrix d_shifted_points,
    Matrix d_new_shift, Matrix d_denominator, Matrix d_mean_shift_vector, double **kernel_matrix,
    double **original_points, double ***new_shift, double ***mean_shift_vector){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)(max_block_size / 2);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = 2;
        dimGrid.x = (d_denominator.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        shift_points_kernel<<<dimGrid, dimBlock>>>(d_original_points, d_kernel_matrix, d_shifted_points,
            d_new_shift, d_denominator, d_mean_shift_vector);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);

    if (first_iter && params.verbose){
        printf("shift_points_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMemcpy(&((*new_shift)[0][0]), d_new_shift.elements
        , size, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(&((*mean_shift_vector)[0][0]), d_mean_shift_vector.elements
        , size, hipMemcpyDeviceToHost) );
}

void free_device_memory(Matrix d_original_points, Matrix d_kernel_matrix, Matrix d_denominator,
    Matrix d_new_shift){
    // frees all memory previously allocated in device
    gpuErrchk( hipFree(d_original_points.elements) );
    gpuErrchk( hipFree(d_kernel_matrix.elements) );
    //gpuErrchk( hipFree(d_shifted_points.elements) );
    gpuErrchk( hipFree(d_denominator.elements) );
    gpuErrchk( hipFree(d_new_shift.elements) );
}

double calculateDistance(double *y, double *x){
    double sum = 0, dif;
    for (int i=0; i<DIMENSIONS; i++){
        dif = y[i]-x[i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);
    return distance;
}

double **alloc_2d_double(int rows, int cols) {
    double *data = (double *) malloc(rows*cols*sizeof(double));
    double **array = (double **) malloc(rows*sizeof(double*));
    for (int i=0; i<rows; i++)
        array[i] = &(data[cols*i]);
    return array;
}

void duplicate(double **source, int rows, int cols, double ***dest){
    for (int i=0; i<rows; i++){
        for (int j=0; j<cols; j++){
            (*dest)[i][j] = source[i][j];
        }
    }
}

void print_matrix(double **array, int rows, int cols){
    for (int i=0; i<cols; i++){
        for (int j=0; j<rows; j++){
            printf("%f ", array[j][i]);
        }
        printf("\n");
    }
}

void save_matrix(double **matrix, int iteration){
    char filename[50];
    snprintf(filename, sizeof(filename), "%s%d", "../output/output_", iteration);
    FILE *file;
    file = fopen(filename, "w");
    for (int rows=0; rows<NUMBER_OF_POINTS; ++rows){
        for (int cols=0; cols<DIMENSIONS; ++cols){
            fprintf(file, "%f", matrix[rows][cols]);
            if (cols != DIMENSIONS - 1){
                fprintf(file, ",");
            }
        }
        fprintf(file, "\n");
    }
}