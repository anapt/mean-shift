#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <sys/time.h>

#include <hipblas.h>

#include "meanshift_utils.h"
#include "meanshift_gpu_utils.h"

hipDeviceProp_t device_properties;

struct timeval start_w_time, end_w_time;
double seq;

//Based on:
//          https://www.cs.virginia.edu/~csadmin/wiki/index.php/CUDA_Support/Choosing_a_GPU
void set_GPU(){
    int devices_count = 0, max_multiprocessors = 0, max_device = 0;

    // gets devices count checking for errors like no devices or no drivers to check for
    // devices available
    gpuErrchk( hipGetDeviceCount(&devices_count) );
    for(int device_index = 0; device_index < devices_count; ++device_index){
        // gets current index device's properties
        hipDeviceProp_t this_device_properties;
        gpuErrchk( hipGetDeviceProperties(&this_device_properties, device_index) );

        // stores best available device's index
        // only devices with compute capability >= 2.0 are able to run the code
        if (max_multiprocessors < this_device_properties.multiProcessorCount
            && this_device_properties.major >= 2 && this_device_properties.minor >= 0){
            // stores devices properties for later use
            device_properties = this_device_properties;
            max_multiprocessors = this_device_properties.multiProcessorCount;
            max_device = device_index;
        }
    }
    // sets the device
    gpuErrchk( hipSetDevice(max_device) );
    if (params.verbose){
        printf("Device chosen is \"%s\"\n"
            "Device has %d multi processors and compute capability %d.%d\n"
            "Max threads per block supported are %d\n\n"
            , device_properties.name
            , device_properties.multiProcessorCount, device_properties.major, device_properties.minor
            , device_properties.maxThreadsPerBlock);
    }
}

int meanshift(double **original_points, double ***shifted_points, int deviation){
    // host variables
    int size = 0;
    static int recursion = 0;
    static double **kernel_matrix, **mean_shift_vector, w_memcpy_time;
    double **new_shift, current_norm = 0, tmp_w_memcpy_time;
    bool is_first_recursion = false;

    // device variables
    static Matrix d_original_points, d_shifted_points, d_kernel_matrix, d_denominator,
        d_mean_shift_vector;
    Matrix d_new_shift;

    // allocates memory and copies original points on first recursion
    if (recursion == 0 || (*shifted_points) == NULL){
        is_first_recursion = true;
        // allocates memory for shifted points array and copies original points into it
        (*shifted_points) = alloc_double(NUMBER_OF_POINTS, DIMENSIONS);
        duplicate(original_points, NUMBER_OF_POINTS, DIMENSIONS, shifted_points);

        // allocates memory for mean shift vector
        mean_shift_vector = alloc_double(NUMBER_OF_POINTS, DIMENSIONS);
        // initializes elements of mean_shift_vector to inf
        for (int i=0;i<NUMBER_OF_POINTS;i++){
            for (int j=0;j<DIMENSIONS;j++){
                mean_shift_vector[i][j] = DBL_MAX;
            }
        }

        // allocates memory for kernel_matrix
        kernel_matrix = alloc_double(NUMBER_OF_POINTS, NUMBER_OF_POINTS);

        // tic
        gettimeofday (&start_w_time, NULL);

        // allocates corresponding memory in device
        init_device_memory(original_points, *shifted_points, &d_original_points, &d_shifted_points,
            &d_kernel_matrix, &d_denominator, &d_mean_shift_vector);
        // toc
        gettimeofday (&end_w_time, NULL);
        seq = (double)((end_w_time.tv_usec - start_w_time.tv_usec)
            / 1.0e6 + end_w_time.tv_sec - start_w_time.tv_sec);

        if (params.verbose){
            printf("Device memory allocation wall clock time = %f\n\n", seq);
        }
    }

    // finds pairwise distance matrix (inside radius)
    // [I, D] = rangesearch(x,y,h);
    calculate_kernel_matrix(d_shifted_points, d_original_points, d_kernel_matrix, deviation,
        &kernel_matrix, &tmp_w_memcpy_time);
    w_memcpy_time += tmp_w_memcpy_time;

    // calculates denominator
    calculate_denominator(d_kernel_matrix, d_denominator);

    // creates new y vector
    // allocates memory in every recursion
    new_shift = alloc_double(NUMBER_OF_POINTS, DIMENSIONS);
    // allocates corresponding memory in device
    d_new_shift.width = DIMENSIONS;
    d_new_shift.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_new_shift.elements), size) );

    shift_points(d_kernel_matrix, d_original_points, d_shifted_points, d_new_shift, d_denominator,
        d_mean_shift_vector, kernel_matrix, original_points, &new_shift, &mean_shift_vector,
        &tmp_w_memcpy_time);
    w_memcpy_time += tmp_w_memcpy_time;

    // frees previously shifted points, they're now garbage
    free((*shifted_points)[0]);
    gpuErrchk( hipFree(d_shifted_points.elements) );
    // updates shifted points pointer to the new array address
    shifted_points = &new_shift;
    d_shifted_points.elements = d_new_shift.elements;

    if (params.display){
        save_matrix((*shifted_points), recursion);
    }

    // calculates norm of the new mean shift vector in GPU using "cuBlas" library function
    // TODO REPLACE WITH KERNEL NORM
//    hipblasHandle_t handle;
//    hipblasStatus_t cublas_status = hipblasCreate(&handle);
//    if (cublas_status != HIPBLAS_STATUS_SUCCESS){
//        exit(cublas_status);
//    }
//    cublas_status = hipblasDnrm2(handle, NUMBER_OF_POINTS * DIMENSIONS, d_mean_shift_vector.elements,
//        1, &current_norm);
//    if (cublas_status != HIPBLAS_STATUS_SUCCESS){
//        exit(cublas_status);
//    }
//    cublas_status = hipblasDestroy(handle);
//    if (cublas_status != HIPBLAS_STATUS_SUCCESS){
//        exit(cublas_status);
//    }
    calculate_norm(d_mean_shift_vector, &current_norm);


    if (params.verbose){
        printf("Recursion n. %d, error\t%f \n", recursion, current_norm);
    }

    // recurses until convergence
    if (current_norm > params.epsilon) {
        ++recursion;
        meanshift(original_points, shifted_points, deviation);
    }

    if (is_first_recursion){
        if (params.verbose){
            printf("\nCopying between device and host wall clock time = %f\n", w_memcpy_time);
        }

        // cleans up allocations
        free(mean_shift_vector[0]);
        free(mean_shift_vector);
        free(kernel_matrix[0]);
        free(kernel_matrix);

        free_device_memory(d_original_points, d_kernel_matrix, d_denominator, d_shifted_points);
    }

    return recursion;
}

void init_device_memory(double **original_points, double **shifted_points,
    Matrix *d_original_points, Matrix *d_shifted_points, Matrix *d_kernel_matrix,
    Matrix *d_denominator, Matrix *d_mean_shift_vector){
    int size;

    // allocates memory for original_points in GPU and copies the array
    d_original_points->width = DIMENSIONS;
    d_original_points->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_original_points->elements), size) );
    gpuErrchk( hipMemcpy(d_original_points->elements, &(original_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for shifted_points in GPU and copies the array
    d_shifted_points->width = DIMENSIONS;
    d_shifted_points->height = NUMBER_OF_POINTS;
    size = DIMENSIONS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_shifted_points->elements), size) );
    gpuErrchk( hipMemcpy(d_shifted_points->elements, &(shifted_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for kernel_matrix in GPU
    d_kernel_matrix->width = NUMBER_OF_POINTS;
    d_kernel_matrix->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_kernel_matrix->elements), size) );

    // allocates memory for denominator in GPU
    d_denominator->width = 1;
    d_denominator->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_denominator->elements), size) );

    // allocates memory for mean_shift_vector in GPU
    d_mean_shift_vector->width = DIMENSIONS;
    d_mean_shift_vector->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_mean_shift_vector->elements), size) );
}

void calculate_kernel_matrix(Matrix d_shifted_points, Matrix d_original_points,
    Matrix d_kernel_matrix, double deviation, double ***kernel_matrix, double *w_memcpy_time){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)sqrt(max_block_size);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = requested_block_size;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = (d_kernel_matrix.width + dimBlock.y - 1) / dimBlock.y;

        calculate_kernel_matrix_kernel<<<dimGrid, dimBlock>>>(d_shifted_points, d_original_points
            , deviation, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_kernel_matrix_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);

    // tic
    gettimeofday (&start_w_time, NULL);

    gpuErrchk( hipMemcpy(&((*kernel_matrix)[0][0]), d_kernel_matrix.elements
        , size, hipMemcpyDeviceToHost) );

    // toc
    gettimeofday (&end_w_time, NULL);
    *w_memcpy_time = (double)((end_w_time.tv_usec - start_w_time.tv_usec)
        / 1.0e6 + end_w_time.tv_sec - start_w_time.tv_sec);
}

void calculate_denominator(Matrix d_kernel_matrix, Matrix d_denominator){
    static bool first_iter = true;
    // gets max block size supported from the device
    static int requested_block_size = device_properties.maxThreadsPerBlock;
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = 1;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        denominator_kernel<<<dimGrid, dimBlock>>>(d_denominator, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_denominator called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }
}

void shift_points(Matrix d_kernel_matrix, Matrix d_original_points, Matrix d_shifted_points,
                  Matrix d_new_shift, Matrix d_denominator, Matrix d_mean_shift_vector, double **kernel_matrix,
                  double **original_points, double ***new_shift,
                  double ***mean_shift_vector, double *w_memcpy_time){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)(max_block_size / d_new_shift.width);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = d_new_shift.width;
        dimGrid.x = (d_denominator.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        shift_points_kernel<<<dimGrid, dimBlock>>>(d_original_points, d_kernel_matrix, d_shifted_points,
            d_new_shift, d_denominator, d_mean_shift_vector);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);

    if (first_iter && params.verbose){
        printf("shift_points_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);

    // tic
    gettimeofday (&start_w_time, NULL);

    gpuErrchk( hipMemcpy(&((*new_shift)[0][0]), d_new_shift.elements
        , size, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(&((*mean_shift_vector)[0][0]), d_mean_shift_vector.elements
        , size, hipMemcpyDeviceToHost) );

    // toc
    gettimeofday (&end_w_time, NULL);
    *w_memcpy_time = (double)((end_w_time.tv_usec - start_w_time.tv_usec)
        / 1.0e6 + end_w_time.tv_sec - start_w_time.tv_sec);
}

void calculate_norm(Matrix d_mean_shift_vector, double *current_norm){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)(max_block_size / d_mean_shift_vector.width);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = 1;
        dimGrid.x = (d_mean_shift_vector.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        norm<<<dimGrid, dimBlock>>>(d_mean_shift_vector, current_norm);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);

    if (first_iter && params.verbose){
        printf("norm_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
}

void free_device_memory(Matrix d_original_points, Matrix d_kernel_matrix, Matrix d_denominator,
    Matrix d_shifted_points){
    // frees all memory previously allocated in device
    gpuErrchk( hipFree(d_original_points.elements) );
    gpuErrchk( hipFree(d_kernel_matrix.elements) );
    gpuErrchk( hipFree(d_denominator.elements) );
    gpuErrchk( hipFree(d_shifted_points.elements) );
}