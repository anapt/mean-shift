#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <sys/time.h>

#include <hipblas.h>

#include "meanshift_utils.h"
#include "meanshift_gpu_utils.h"

hipDeviceProp_t device_properties;

struct timeval start, end;
double seq;

//Based on:
//          https://www.cs.virginia.edu/~csadmin/wiki/index.php/CUDA_Support/Choosing_a_GPU
void set_GPU(){
    int devices_count = 0, max_multiprocessors = 0, max_device = 0;

    // gets devices count checking for errors like no devices or no drivers to check for
    // devices available
    gpuErrchk( hipGetDeviceCount(&devices_count) );
    for(int device_index = 0; device_index < devices_count; ++device_index){
        // gets current index device's properties
        hipDeviceProp_t this_device_properties;
        gpuErrchk( hipGetDeviceProperties(&this_device_properties, device_index) );

        // stores best available device's index
        // only devices with compute capability >= 2.0 are able to run the code
        if (max_multiprocessors < this_device_properties.multiProcessorCount
            && this_device_properties.major >= 2 && this_device_properties.minor >= 0){
            // stores devices properties for later use
            device_properties = this_device_properties;
            max_multiprocessors = this_device_properties.multiProcessorCount;
            max_device = device_index;
        }
    }
    // sets the device
    gpuErrchk( hipSetDevice(max_device) );
    if (params.verbose){
        printf("Device chosen is \"%s\"\n"
            "Device has %d multi processors and compute capability %d.%d\n"
            "Max threads per block supported are %d\n\n"
            , device_properties.name
            , device_properties.multiProcessorCount, device_properties.major, device_properties.minor
            , device_properties.maxThreadsPerBlock);
    }
}

int meanshift(double **original_points, double ***shifted_points, int deviation
    , parameters *opt){
    // host variables
    int size = 0;
    static int iteration = 0;
    static double **kernel_matrix, **mean_shift_vector;
    double **new_shift, current_norm = 0;

    // device variables
    static Matrix d_original_points, d_shifted_points, d_kernel_matrix, d_denominator,
        d_mean_shift_vector;
    Matrix d_new_shift;

    // allocates memory and copies original points on first iteration
    if (iteration == 0 || (*shifted_points) == NULL){
        // allocates memory for shifted points array and copies original points into it
        (*shifted_points) = alloc_double(NUMBER_OF_POINTS, DIMENSIONS);
        duplicate(original_points, NUMBER_OF_POINTS, DIMENSIONS, shifted_points);

        // allocates memory for mean shift vector
        mean_shift_vector = alloc_double(NUMBER_OF_POINTS, DIMENSIONS);
        // initializes elements of mean_shift_vector to inf
        for (int i=0;i<NUMBER_OF_POINTS;i++){
            for (int j=0;j<DIMENSIONS;j++){
                mean_shift_vector[i][j] = DBL_MAX;
            }
        }

        // allocates memory for kernel_matrix
        kernel_matrix = alloc_double(NUMBER_OF_POINTS, NUMBER_OF_POINTS);

        // tic
        gettimeofday (&start, NULL);

        // allocates corresponding memory in device
        init_device_memory(original_points, *shifted_points, &d_original_points, &d_shifted_points,
            &d_kernel_matrix, &d_denominator, &d_mean_shift_vector);
        // toc
        gettimeofday (&end, NULL);
        seq = (double)((end.tv_usec - start.tv_usec)/1.0e6 + end.tv_sec - start.tv_sec);

//        printf("%s wall clock time = %f\n","Device memory allocation", seq);
        // to create output data file
        printf("%f ", seq);
    }

    // finds pairwise distance matrix (inside radius)
    // [I, D] = rangesearch(x,y,h);
    calculate_kernel_matrix(d_shifted_points, d_original_points, d_kernel_matrix, deviation,
        &kernel_matrix);

    // calculates denominator
    calculate_denominator(d_kernel_matrix, d_denominator);

    // creates new y vector
    // allocates memory in every recursion
    new_shift = alloc_double(NUMBER_OF_POINTS, DIMENSIONS);
    // allocates corresponding memory in device
    d_new_shift.width = DIMENSIONS;
    d_new_shift.height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_new_shift.elements), size) );

    shift_points(d_kernel_matrix, d_original_points, d_shifted_points, d_new_shift, d_denominator,
        d_mean_shift_vector, kernel_matrix, original_points, &new_shift, &mean_shift_vector);

    // frees previously shifted points, they're now garbage
    free((*shifted_points)[0]);
    // updates shifted points pointer to the new array address
    shifted_points = &new_shift;
    d_shifted_points.elements = d_new_shift.elements;

    if (params.display){
        save_matrix((*shifted_points), iteration);
    }

    // calculates norm of the new mean shift vector in GPU using "cuBlas" library function
    hipblasHandle_t handle;
    hipblasStatus_t cublas_status = hipblasCreate(&handle);
    if (cublas_status != HIPBLAS_STATUS_SUCCESS){
        exit(cublas_status);
    }
    cublas_status = hipblasDnrm2(handle, NUMBER_OF_POINTS * DIMENSIONS, d_mean_shift_vector.elements,
        1, &current_norm);
    if (cublas_status != HIPBLAS_STATUS_SUCCESS){
        exit(cublas_status);
    }
    cublas_status = hipblasDestroy(handle);
    if (cublas_status != HIPBLAS_STATUS_SUCCESS){
        exit(cublas_status);
    }

    if (params.verbose){
        printf("Iteration n. %d, error\t%f \n", iteration, current_norm);
    }

    // iterates until convergence
    if (current_norm > opt->epsilon) {
        ++iteration;
        meanshift(original_points, shifted_points, deviation, opt);
    }

    if (iteration == 0){
        // cleans up allocations
        free(mean_shift_vector[0]);
        free(mean_shift_vector);
        free(kernel_matrix[0]);
        free(kernel_matrix);

        free_device_memory(d_original_points, d_kernel_matrix, d_denominator, d_new_shift);
    }

    return iteration;
}

void init_device_memory(double **original_points, double **shifted_points,
    Matrix *d_original_points, Matrix *d_shifted_points, Matrix *d_kernel_matrix,
    Matrix *d_denominator, Matrix *d_mean_shift_vector){
    int size;

    // allocates memory for original_points in GPU and copies the array
    d_original_points->width = DIMENSIONS;
    d_original_points->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_original_points->elements), size) );
    gpuErrchk( hipMemcpy(d_original_points->elements, &(original_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for shifted_points in GPU and copies the array
    d_shifted_points->width = DIMENSIONS;
    d_shifted_points->height = NUMBER_OF_POINTS;
    size = DIMENSIONS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_shifted_points->elements), size) );
    gpuErrchk( hipMemcpy(d_shifted_points->elements, &(shifted_points[0][0])
        , size, hipMemcpyHostToDevice) );

    // allocates memory for kernel_matrix in GPU
    d_kernel_matrix->width = NUMBER_OF_POINTS;
    d_kernel_matrix->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_kernel_matrix->elements), size) );

    // allocates memory for denominator in GPU
    d_denominator->width = 1;
    d_denominator->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_denominator->elements), size) );

    // allocates memory for mean_shift_vector in GPU
    d_mean_shift_vector->width = DIMENSIONS;
    d_mean_shift_vector->height = NUMBER_OF_POINTS;
    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);
    gpuErrchk( hipMalloc(&(d_mean_shift_vector->elements), size) );
}

void calculate_kernel_matrix(Matrix d_shifted_points, Matrix d_original_points,
    Matrix d_kernel_matrix, double deviation, double ***kernel_matrix){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)sqrt(max_block_size);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = requested_block_size;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = (d_kernel_matrix.width + dimBlock.y - 1) / dimBlock.y;

        calculate_kernel_matrix_kernel<<<dimGrid, dimBlock>>>(d_shifted_points, d_original_points
            , deviation, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_kernel_matrix_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * NUMBER_OF_POINTS * sizeof(double);

    // tic
    gettimeofday (&start, NULL);
    gpuErrchk( hipMemcpy(&((*kernel_matrix)[0][0]), d_kernel_matrix.elements
        , size, hipMemcpyDeviceToHost) );

    // toc
    gettimeofday (&end, NULL);
    seq = (double)((end.tv_usec - start.tv_usec)/1.0e6 + end.tv_sec - start.tv_sec);

//    printf("%s wall clock time = %f\n","Copying from device to host", seq);
    // to create output data file
        printf("%f ", seq);
}

void calculate_denominator(Matrix d_kernel_matrix, Matrix d_denominator){
    static bool first_iter = true;
    // gets max block size supported from the device
    static int requested_block_size = device_properties.maxThreadsPerBlock;
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = 1;
        dimGrid.x = (d_kernel_matrix.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        denominator_kernel<<<dimGrid, dimBlock>>>(d_denominator, d_kernel_matrix);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);
    
    if (first_iter && params.verbose){
        printf("calculate_denominator called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

}

void shift_points(Matrix d_kernel_matrix, Matrix d_original_points, Matrix d_shifted_points,
    Matrix d_new_shift, Matrix d_denominator, Matrix d_mean_shift_vector, double **kernel_matrix,
    double **original_points, double ***new_shift, double ***mean_shift_vector){
    int size;
    static bool first_iter = true;
    // gets max block size supported from the device
    static int max_block_size = device_properties.maxThreadsPerBlock;
    static int requested_block_size = (int)(max_block_size / d_new_shift.width);
    bool block_size_too_big = true;

    dim3 dimBlock;
    dim3 dimGrid;
    do {
        dimBlock.x = requested_block_size;
        dimBlock.y = d_new_shift.width;
        dimGrid.x = (d_denominator.height + dimBlock.x - 1) / dimBlock.x;
        dimGrid.y = 1;

        shift_points_kernel<<<dimGrid, dimBlock>>>(d_original_points, d_kernel_matrix, d_shifted_points,
            d_new_shift, d_denominator, d_mean_shift_vector);
        if (hipGetLastError() != hipSuccess){
            --requested_block_size;
        } else {
            block_size_too_big = false;
            gpuErrchk( hipDeviceSynchronize() );
        }
    } while(block_size_too_big);

    if (first_iter && params.verbose){
        printf("shift_points_kernel called with:\n");
        printf("dimBlock.x = %d, dimBlock.y = %d\n", dimBlock.x, dimBlock.y);
        printf("dimGrid.x = %d, dimGrid.y = %d\n\n", dimGrid.x, dimGrid.y);
        first_iter = false;
    }

    size = NUMBER_OF_POINTS * DIMENSIONS * sizeof(double);

    // tic
    gettimeofday (&start, NULL);

    gpuErrchk( hipMemcpy(&((*new_shift)[0][0]), d_new_shift.elements
        , size, hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(&((*mean_shift_vector)[0][0]), d_mean_shift_vector.elements
        , size, hipMemcpyDeviceToHost) );

    // toc
    gettimeofday (&end, NULL);
    seq = (double)((end.tv_usec - start.tv_usec)/1.0e6 + end.tv_sec - start.tv_sec);

//    printf("%s wall clock time = %f\n","Copying from device to host", seq);
    // to create output data file
        printf("%f ", seq);

}

void free_device_memory(Matrix d_original_points, Matrix d_kernel_matrix, Matrix d_denominator,
    Matrix d_new_shift){
    // frees all memory previously allocated in device
    gpuErrchk( hipFree(d_original_points.elements) );
    gpuErrchk( hipFree(d_kernel_matrix.elements) );
    gpuErrchk( hipFree(d_denominator.elements) );
    gpuErrchk( hipFree(d_new_shift.elements) );
}