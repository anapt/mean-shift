
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 11
#define M 3

__global__ void kernel(float * d_matrix, size_t pitch) {
    for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < N; j += blockDim.y * gridDim.y) {
        float* row_d_matrix = (float*)((char*)d_matrix + j*pitch);
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M; i += blockDim.x * gridDim.x) {
            row_d_matrix[i] = (j * M + i) + (j * M + i);
        }
    }
}

void verify(float *h, float *d, int size) {
    for (int i = 0; i < size; i++) {
        assert(h[i] == d[i]);
    }
    printf("Results match\n");
}

int main() {

    float *h_matrix;
    float *d_matrix;
    float *dc_matrix;

    h_matrix = (float *) malloc(M * N * sizeof(float));
    dc_matrix = (float *) malloc(M * N * sizeof(float));

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < M; i++) {
            h_matrix[j * M + i] = (j * M + i) + (j * M + i);
        }
    }

    size_t pitch;
    hipMallocPitch(&d_matrix, &pitch, M * sizeof(float), N);

    dim3 grid(1, 1, 1);
    dim3 block(3, 3, 1);

    kernel<<<grid, block>>>(d_matrix, pitch);

    hipMemcpy2D(dc_matrix, M * sizeof(float), d_matrix, pitch, M * sizeof(float), N, hipMemcpyDeviceToHost);

    verify(h_matrix, dc_matrix, M * N);

    free(h_matrix);
    hipFree(d_matrix);
    free(dc_matrix);
}
