#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <stdbool.h>
#include <math.h>

#define X "data/X.bin"
#define L "data/L.bin"
#define COLUMNS     2
#define ROWS        600

struct parameters {
    double epsilon;
    bool verbose;
    bool display;
};

double **alloc_2d_double(int rows, int cols);
double **duplicate(double **a, double **b, int rows, int cols);
void meanshift(double **x, int h, struct parameters *opt);
double norm(double ** m, int rows, int cols);
void multiply(double ** matrix1, double ** matrix2, double ** output);
double calculateDistance(double *, double *);
void print_matrix(double ** array, int rows, int cols);


struct timeval startwtime, endwtime;
double seq_time;

int main(int argc, char **argv){

//    if (argc<2){
//        printf("%s\n", "Specify the k");
//        return 1;
//    }
//    = atoi(argv[1]);  // the k-parameter


    FILE *f;
//    f = fopen(X, "rb");
//    fseek(f, 0L, SEEK_END);
//    long int pos = ftell(f);
//    fclose(f);
//    int elements = pos / sizeof(double);  // number of total elements (points*dimension)
//    int points = elements/COLUMNS;
//    //printf("points : %d \n", points);
    f = fopen(X, "rb");
    double ** vectors;
    vectors = alloc_2d_double(ROWS, COLUMNS);
    for (int i=0; i<ROWS; i++){
        int out = fread(vectors[i], sizeof(double), COLUMNS, f);
    }
    //printf("test : %f \n", vectors[0][0]);
    //printf("test : %f \n", vectors[ROWS-1][COLUMNS-1]);

    // initializing file that will contain the labels (train)
    f = fopen(L, "rb");
    // NOTE : Labels were classified as <class 'numpy.uint8'>
    // variables of type uint8 are stored as 1-byte (8-bit) unsigned integers
    fseek(f, 0L, SEEK_END);
    long int pos = ftell(f);
    rewind(f);
    //printf("position : %ld \n", pos);
    int label_elements = pos/ sizeof(char);
    char *labels = (char*)malloc(label_elements* sizeof(char));
    fseek(f, 0L, SEEK_SET);
    int out = fread(labels, sizeof(char), label_elements, f);
    fclose(f);

    // MEAN SHIFT OPTIONS
    int h = 1;
    struct parameters params;
    params.epsilon = 0.0001;
    params.verbose = false;
    params.display = false;
    struct parameters *opt;
    opt = &params;

    // tic
    gettimeofday (&startwtime, NULL);

    meanshift(vectors, h, opt);

    // toc
    gettimeofday (&endwtime, NULL);
    seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
    printf("%s wall clock time = %f\n","Mean Shift", seq_time);

    //TODO write output points to file -> plot later

}

void meanshift(double **x, int h, struct parameters *opt){

    double **y;
    y = alloc_2d_double(ROWS, COLUMNS);
    y = duplicate(x, y, ROWS, COLUMNS);

    // mean shift vectors
    double **m;
    m = alloc_2d_double(ROWS, COLUMNS);
    // initialize elements of m to inf
    for (int i=0;i<ROWS;i++){
        for (int j=0;j<COLUMNS;j++){
            m[i][j] = DBL_MAX;
        }
    }

    // initialize iteration counter
    int iter = 0;

    // printf("%f \n", opt->epsilon);

    /** iterate until convergence **/
    // printf("norm : %f \n", norm(m, ROWS, COLUMNS));
    /** allocate memory **/
    double ** W = alloc_2d_double(ROWS, ROWS);
    double * l = malloc(ROWS * sizeof(double));

    double * d_W;
    hipMalloc(&d_W, ROWS * ROWS * sizeof(double));
    double * d_I;
    hipMalloc(&d_I, ROWS * sizeof(double));
    double * d_y_new;
    hipMalloc(&d_y_new, ROWS * COLUMNS * sizeof(double));

    double * d_y;
    hipMalloc(&d_y, ROWS * COLUMNS * sizeof(double));
    double * d_m;
    hipMalloc(&d_m, ROWS * COLUMNS * sizeof(double));

    //Copy vectors from host memory to device memory
    hipMemcpy(d_y, y, ROWS * COLUMNS * sizeof(double), hipMemcpyHostToDevice);
    // y[i][j] == d_y[COLUMNS*i + j]
    hipMemcpy(d_m, m, ROWS * COLUMNS * sizeof(double), hipMemcpyHostToDevice);


    while (norm(m, ROWS, COLUMNS) > opt->epsilon) {
        iter = iter +1;
        // find pairwise distance matrix (inside radius)
        /** allocate memory for inside iteration arrays **/
        double ** W = alloc_2d_double(ROWS, ROWS);
        double * l = malloc(ROWS * sizeof(double));
        // [I, D] = rangesearch(x,y,h);
        for (int i=0; i<ROWS; i++){
            for (int j=0; j<ROWS; j++){
                double dist = calculateDistance(y[i],x[j]);

                // 2sparse matrix
                if (dist < h){
                    W[i][j] = dist;
                    //printf("%f \n", W[i][j]);
                }else{
                    W[i][j] = 0;
                }
            }
        }


        // for each element of W (x) do x^2
        // size of W is [600 600]
        // W is a sparse matrix -> apply to non-zero elements
        for (int i=0; i<ROWS; i++){
            double sum =0;
            for (int j=0; j < ROWS; j++){
                if (W[i][j] != 0){
                    W[i][j] = W[i][j]*W[i][j];
                    // compute kernel matrix
                    // apply function to non zero elements of a sparse matrix
                    double pow = ((-1)*(W[i][j]))/(2*(h*h));
                    W[i][j] = exp(pow);
                }
                // make sure diagonal elements are 1
                if (i==j){
                    W[i][j] = W[i][j] +1;
                }
                // calculate sum(W,2)
                sum = sum + W[i][j];
            }
            /** l array is correct**/
            l[i] = sum;
            // printf("l[%d] : %f \n", i, l[i]);
        }
        /** W is correct**/
        //print_matrix(W, ROWS, ROWS);


        // create new y vector
        double** y_new = alloc_2d_double(ROWS, COLUMNS);

        multiply(W, x, y_new);
        /** y_new is CORRECT **/
        // print_matrix(y_new, ROWS, COLUMNS);
        // divide element-wise
        for (int i=0; i<ROWS; i++){
            for (int j=0; j<COLUMNS; j++){
                y_new[i][j] = y_new[i][j] / l[i];
            }
        }

        // calculate mean-shift vector
        for (int i=0; i<ROWS; i++){
            for (int j=0; j<COLUMNS; j++){
                m[i][j] = y_new[i][j] - y[i][j];

                // update y
                y[i][j] = y_new[i][j];
            }
        }

        printf("Iteration n. %d, error %f \n", iter, norm(m, ROWS, COLUMNS));
        // TODO maybe keep y for live display later?
    };



}

// allocates a 2d array in continuous memory positions
double **alloc_2d_double(int rows, int cols) {
    double *data = (double *)malloc(rows*cols*sizeof(double));
    double **array= (double **)malloc(rows*sizeof(double*));
    for (int i=0; i<rows; i++)
        array[i] = &(data[cols*i]);
    return array;
}

// copy the values of a 2d double array to another
double **duplicate(double **a, double **b, int rows, int cols){
    for (int i=0;i<rows;i++){
        for (int j=0;j<cols;j++){
            b[i][j] = a[i][j];
        }
    }
    return b;
}

// TODO check why there's is a difference in the norm calculate in matlab
double norm(double ** m, int rows, int cols){
    double sum=0, a=0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            a = m[i][j] * m[i][j];
            sum = sum + a;
        }
    }
    double norm = sqrt(sum);
    return norm;
}

double calculateDistance(double *y, double *x){
    double sum = 0, dif;
    for (int i=0;i<COLUMNS;i++){
        dif = y[i]-x[i];
        sum += dif * dif;
    }
    double distance = sqrt(sum);
    return distance;
}

void multiply(double ** matrix1, double ** matrix2, double ** output){
    // W dims are ROWS ROWS and x dims are ROWS COLUMNS

    int i, j, k;
    for (i=0; i<ROWS; i++){
        for (j=0; j<COLUMNS; j++){
            output[i][j] = 0;
            for (k=0; k<ROWS; k++){
                output[i][j] += matrix1[i][k] * matrix2[k][j];
            }
        }
    }
}

void print_matrix(double ** array, int rows, int cols){
    for (int i=0; i<cols; i++){
        for (int j=0; j<rows; j++){
            printf("%f ", array[j][i]);
        }
        printf("\n");
    }
}

__global__ void iteration (double* W, double epsilon){
    // TODO check if they also need cudamalloc
    // todo find how to keep counter
    int iter;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    while (norm > epsilon){
        // TODO ITERATION
        iter = iter +1;
        // find pairwise distance matrix (inside radius)
        /** allocate memory for inside iteration arrays **/
        // TODO ALLOCATE MEMORY BEFORE CALLING KERNEL
//        double ** W = alloc_2d_double(ROWS, ROWS);
//        double * l = malloc(ROWS * sizeof(double));
        // [I, D] = rangesearch(x,y,h);
        for (int i=0; i<ROWS; i++){
            for (int j=0; j<ROWS; j++){
                // TODO REFACTOR CALCULATE DISTANCE
                double dist = calculateDistance(y[i],x[j]);

                // 2sparse matrix
                if (dist < h){
                    W[i][j] = dist;
                    //printf("%f \n", W[i][j]);
                }else{
                    W[i][j] = 0;
                }
            }
        }


        // for each element of W (x) do x^2
        // size of W is [600 600]
        // W is a sparse matrix -> apply to non-zero elements
        for (int i=0; i<ROWS; i++){
            double sum =0;
            for (int j=0; j < ROWS; j++){
                if (W[i][j] != 0){
                    W[i][j] = W[i][j]*W[i][j];
                    // compute kernel matrix
                    // apply function to non zero elements of a sparse matrix
                    double pow = ((-1)*(W[i][j]))/(2*(h*h));
                    W[i][j] = exp(pow);
                }
                // make sure diagonal elements are 1
                if (i==j){
                    W[i][j] = W[i][j] +1;
                }
                // calculate sum(W,2)
                sum = sum + W[i][j];
            }
            /** l array is correct**/
            l[i] = sum;
            // printf("l[%d] : %f \n", i, l[i]);
        }
        /** W is correct**/
        //print_matrix(W, ROWS, ROWS);


        // create new y vector
        double** y_new = alloc_2d_double(ROWS, COLUMNS);

        multiply(W, x, y_new);
        /** y_new is CORRECT **/
        // print_matrix(y_new, ROWS, COLUMNS);
        // divide element-wise
        for (int i=0; i<ROWS; i++){
            for (int j=0; j<COLUMNS; j++){
                y_new[i][j] = y_new[i][j] / l[i];
            }
        }

        // calculate mean-shift vector
        for (int i=0; i<ROWS; i++){
            for (int j=0; j<COLUMNS; j++){
                m[i][j] = y_new[i][j] - y[i][j];

                // update y
                y[i][j] = y_new[i][j];
            }
        }

        printf("Iteration n. %d, error %f \n", iter, norm(m, ROWS, COLUMNS));
        // TODO maybe keep y for live display later?
    }
}