#include "hip/hip_runtime.h"

// Host code 
int width = 64, height = 64; 
float* devPtr; 
size_t pitch; 
hipMallocPitch(&devPtr, &pitch, width * sizeof(float), height); 
MyKernel<<<100, 512>>>(devPtr, pitch, width, height); 


// Device code 
__global__ voidMyKernel(float* devPtr, size_t pitch, int width, int height) { 
    for (int r = 0; r < height; ++r) { 
        float* row = (float*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c) {
            float element = row[c]; 
        } 
    } 
}

Read more at: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#ixzz54kTh80mZ 
Follow us: @GPUComputing on Twitter | NVIDIA on Facebook